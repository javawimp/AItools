#include "includes.h"
#include "externs.h"
#include "State.h"

//[w][x][y][z] = [sx*sy*sz]*w+[sy*sz]*x+[sz]*y+z
//[NB_SUMS][NB_DLR][NB_FLAGS * 2][NB_ACTIONS];
#define TABLEINDEX(sum,dlr,flags,action) \
((NB_DLR * NB_FLAGS * 2*NB_ACTIONS)*sum +\
(NB_FLAGS * 2*NB_ACTIONS)*dlr +\
(NB_ACTIONS)*flags + action)

__device__ void Qget(State *s, char action, double *result)
    {
        if (s == NULL) {
		printf("updateQValues encountered null state!\n");
		hipDeviceSynchronize();
		return;
        }
        int sum = s->sum;
        int dlr = s->dealer - 1;
        int flags = s->flags;
        if (verbose>9)
		printf("Qget sum=%d dlr=%d flags=%d action=%d q=%6.3f\n", sum, dlr, flags, action,
			table[TABLEINDEX(sum,dlr,flags,action)]);
        if (sum > (NB_SUMS - 1) || dlr > (NB_DLR - 1) || flags > (NB_FLAGS * 2 - 1) || action >= NB_ACTIONS) {
		//printf("Qget sum=%d dlr=%d flags=%d action=%d\n", sum, dlr, flags, action);
		*result=0.0;
		return;
        }
        //*result = table[sum][dlr][flags][action];
        *result = table[TABLEINDEX(sum,dlr,flags,action)];
    }

__device__ void Qset(State *s, char action, double value)
    {
        if (s == NULL) {
		printf("cannot Qset null state\n");
		hipDeviceSynchronize();
		return;
        }
        int sum = s->sum;
        int dlr = s->dealer - 1;
        int flags = s->flags;
        if (verbose>9) printf("Qset sum=%d dlr=%d flags=%d action=%d q=%6.3f\n", sum, dlr, flags, action, value);
        if (sum > (NB_SUMS - 1) || dlr > (NB_DLR - 1) || flags > (NB_FLAGS * 2 - 1) || action >= NB_ACTIONS) {
		printf("out of table error in Qset: sum=%d dlr=%d flags=%d action=%d\n", sum, dlr, flags, action);
		return;
        }
        //table[sum][dlr][flags][action] = value;
        table[TABLEINDEX(sum,dlr,flags,action)] = value;
}

    // The Q(lambda) �learning algorithm
__device__ void updateQValues(State *s, char act, float rwd, State *next_s, char next_a)
 {
        if (s == NULL) {
		printf("updateQValues encountered null state!\n");
		hipDeviceSynchronize();
		return;
        }
        if (s->terminal == 1) {
		printf("updateQValues encountered terminal state!\n");
		hipDeviceSynchronize();
	}
        double Q_next = 0.0;
	double oldValue = 0.0;
	Qget(next_s, next_a, &oldValue);
        if (next_s != NULL && (next_s->terminal==0)) Q_next = Sarsa_gamma * oldValue;
        Qget(s, act, &oldValue);
        double newValue = oldValue + Sarsa_alpha * (rwd + Q_next - oldValue);
        Qset(s, act, newValue);
 }

__device__ void updateSarsa(State *s)
{
        if (s->terminal==1) return;
        char a_t = s->action;
        float reward = (float) s->bank;
        if (s->child1 != NULL) {
		State *next_s = s->child1;
		char next_a = next_s->action;
		updateQValues(s, a_t, reward, next_s, next_a);
		updateSarsa(s->child1);
        }
        if (s->child2 != NULL) {
		State *next_s = s->child2;
		char next_a = next_s->action;
		updateQValues(s, a_t, reward, next_s, next_a);
		updateSarsa(s->child2);
        }
}
