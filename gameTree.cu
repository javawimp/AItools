#include "includes.h"
#include "externs.h"
#include "bjState.h"

__device__ const char *choice[4] = {"STAND","HIT","DOUBLE","SPLIT"};
__device__ const char *cardName[14] = { "","n A"," 2"," 3"," 4"," 5"," 6"," 7","n 8"," 9"," 10"," J"," Q"," K" };

__device__ void cardString(short ncards,short *cards,char *cardbuf)
{
	cardbuf[0] = '{';
	cardbuf[1] = strcard[DENOM(cards[0])];
    for (int i=1;i<ncards;++i) {
    	cardbuf[2*i] = ',';
    	cardbuf[2*i+1] = strcard[DENOM(cards[i])];
    	cardbuf[2*i+2] = '}';
    	cardbuf[2*i+3] = 0;
    }
}
// pick an action consistent with a state node's flags
__device__ void chooseAction(BJState *s,int *amax)
{
	int tid = threadIdx.x;
    *amax = AC_STAND;
    if (s->terminal) return;
    for (;;) {
        *amax = (int) (hiprand_uniform(&state[tid]) * NB_ACTIONS);
        if (*amax == AC_DOUBLE && (s->flags & OK_DOUBLE) == 0) continue;
        if (*amax == AC_SPLIT && (s->flags & OK_SPLIT) == 0) continue;
        break;
    }
    //printf("choosing %s\n",choice[*amax]);
}

__device__ void nextState(BJState *state, int action, int nextC, BJState **nextP)
{
    bool splittingAces = false;
    BJState next = BJState();
	*nextP = &next;
    next.bet = (action == AC_DOUBLE ? 2 : 1) * state->bet;
    next.dealerUp = state->dealerUp;
    for (int i=0;i<16;++i) next.cards[i] = state->cards[i];
    int ncards = state->ncards;
    if (action == AC_SPLIT) --ncards;
    if (ncards == 1) {
        int denom1 = DENOMV(next.cards[0]);
        splittingAces = (denom1 == 1 && action == AC_SPLIT);
        if (!splittingAces) next.flags |= OK_DOUBLE;
        ////if (denom1 == DENOMV(nextC)) next.flags |= OK_SPLIT;
    }
    next.ncards = ncards;
    if (nextC != 0) {
        next.cards[ncards] = nextC;
        ++next.ncards;
    }
    bool issoft;
    sum(next.ncards, next.cards, &issoft, &next.sum);
    if (issoft) next.flags |= OK_SOFT;
    else if (next.sum > 21) next.terminal = true;
    if (splittingAces || action == AC_STAND || action == AC_DOUBLE) next.terminal = true;
    state->actionTaken = action;
    if (verbose) {
    	char cardbuf[50];
    	cardString(next.ncards,next.cards,cardbuf);
        printf("ac=%d(%s),nc=%d %s sum=%d,flags=%d,bet=%d\n", action, choice[action],
        	next.ncards, cardbuf, next.sum, next.flags, next.bet);
		hipDeviceSynchronize();
    }
}

// recursively gen the game tree under this state node
__device__ void playHand(BJState *s,int *deck)
{
	if (s->terminal) {
		return;
	}
	int action;
	chooseAction(s,&action); // no sarsa learning
	//int action = selectAction(s);
	if (verbose>0) {
		char cardbuf[50];
		cardString(s->ncards,s->cards,cardbuf);
        printf("ac=%d(%s),nc=%d %s sum=%d,flags=%d,bet=%d\n", action, choice[action],
        	s->ncards, cardbuf, s->sum, s->flags, s->bet);
	}
	s->actionTaken = action;
	short nextC=0;
	if (action != AC_STAND) {
		nextCard(deck,&nextC);
		if (verbose>0) printf("Player draws a%s\n",cardName[DENOM(nextC)]);
	}
	BJState *next=NULL;
	nextState(s, action, nextC, &next);
	s->child1 = next;
	next->parent = s;
	// if (next.flags & flags_terminal && next.sum <= 21) ++unbusted;
	if (action == AC_SPLIT) {
		short t = s->cards[0];
		s->cards[0] = s->cards[1];
		nextCard(deck,&nextC);
		nextState(s, action, nextC, &next);
		s->child2 = next;
		next->parent = s;
		s->cards[0] = t;
		// if (next.flags & flags_terminal && next.sum <= 21) ++unbusted;
	}
	if (s->child1 != NULL) playHand(s->child1,deck);
	if (s->child2 != NULL) playHand(s->child2,deck);
}
