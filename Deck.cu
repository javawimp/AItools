#include "includes.h"
#include "externs.h"
#include "State.h"

__device__ void cardString(char ncards, char *cards, char *cardbuf)
{
	cardbuf[0] = '{';
	cardbuf[1] = strcard[DENOM(cards[0])];
	for (int i=1;i<ncards;++i) {
		cardbuf[2*i] = ',';
		cardbuf[2*i+1] = strcard[DENOM(cards[i])];
		cardbuf[2*i+2] = '}';
		cardbuf[2*i+3] = 0;
	}
}

__device__ void initializeDeck() {
	int tid = threadIdx.x;
	HandLedger *h = &MasterLedger[tid];
	for (int c=0; c<52; ++c) h->deck[c] = c+1;
}

__device__ void shuffleDeck() {
	int tid = threadIdx.x;
	HandLedger *h = &MasterLedger[tid];
	unsigned int RAND;
	int temp[52];
	for (int newpos = 51; newpos >= 0; --newpos) {
		RAND = hiprand(&randstate[tid]);
		int oldpos = RAND % (newpos+1);
		temp[newpos] = h->deck[oldpos];
		for (int c = oldpos; c < newpos; ++c) h->deck[c] = h->deck[c+1];
	}
	for (int c = 0; c < 52; ++c) h->deck[c] = temp[c];
/*
	// test the shuffle
	printf("tid %d: ",threadIdx.x);
	for (int c = 0; c< 52; ++c) printf("%2d ",h->deck[c]);
	printf("\n");
	hipDeviceSynchronize();
*/
}

__device__ void dealDeck(char *dealt) {
	int tid = threadIdx.x;
	HandLedger *h = &MasterLedger[tid];
	*dealt = h->deck[h->nextCard++];
}

__device__ void psumDeck(int handnum, char *issoft, char *result) {
	int tid = threadIdx.x;
	HandLedger *h = &MasterLedger[tid];
	int sum = 0;
	int soft = 0;
	for (int c=0; c<h->ncards[handnum]; ++c) {
		int denom = DENOMVAL(h->cards[handnum][c]);
		if (denom == 1) {
			denom = 11;
			++soft;
		}
		sum += denom;
	}
	while (sum > 21 && soft > 0) {
		sum -= 10;
		--soft;
	}
	*issoft = (soft>0);
	*result = (char)sum;
}

__device__ void dsumDeck(char ncards, char *issoft, char *result) {
	int tid = threadIdx.x;
	int sum = 0;
	int soft = 0;
	for (int c=0; c<ncards; ++c) {
		int denom = DENOMVAL(dcards[tid][c]);
		if (denom == 1) {
			denom = 11;
			++soft;
		}
		sum += denom;
	}
	while (sum > 21 && soft > 0) {
		sum -= 10;
		--soft;
	}
	*issoft = (soft>0);
	*result = (char)sum;
}
