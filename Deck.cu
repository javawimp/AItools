#include "includes.h"
#include "externs.h"

__device__ void newDeck(int **deck,hipError_t *st) {
	int tid = threadIdx.x;
	if (verbose>2) printf("iter %d: shuffling\n",tid);
	unsigned int RAND;
	int temp[53];
	int *tdeck;
	*st = hipMalloc((void**)&tdeck, 53 * sizeof(int));
	if (*st != hipSuccess) {
		// compile with -rdc=true
		printf("cannot allocate deck, err = %d\n",*st);
		return;
	}
	int x = 0;
	for (int j=1; j<53; ++j) tdeck[x++] = j;
	tdeck[52] = 0;
	for (int newpos = 51; newpos >= 0; newpos--) {
		RAND = hiprand(&state[tid]);
		int oldpos = RAND % (newpos+1);
		temp[newpos] = tdeck[oldpos];
		for (int j = oldpos; j < newpos; j++) tdeck[j] = tdeck[j+1];
	}
	for (int i = 0; i < 52; ++i) tdeck[i] = temp[i];
/*
	// test the shuffle
	//lock(&mutex);
	printf("tid %d: ",threadIdx.x);
	for (int j = 0; j < 52; ++j) printf("%2d ",tdeck[j]);
	printf("\n");
	hipDeviceSynchronize();
	//unlock(&mutex);
*/
	*deck = tdeck;
}

__device__ void nextCard(int *deck, short *dealt) {
	int *card = deck;
	while (*card==-1) ++card;
	*dealt = *card;
	*card = -1;
	//printf("card %2d\n",*dealt);
}

__device__ void sum(short ncards, short *hand, bool *issoft, short *result) {
	short sum = 0;
	int soft = 0;
	for (int j=0; j<ncards; ++j) {
		int denom = DENOMV(hand[j]);
		if (denom == 1) {
			denom = 11;
			++soft;
		}
		sum += denom;
	}
	while (sum > 21 && soft > 0) {
		sum -= 10;
		--soft;
	}
	*issoft = (soft>0);
	*result = sum;
}
