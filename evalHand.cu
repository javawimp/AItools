#include "includes.h"
#include "externs.h"
#include "State.h"

/*
	this routine descends to the leaf nodes, evaluates the final
	profit/loss of each node representing one hand after any
	splits,and passes the "reward" back up the tree as "contrib".
	"contrib" will be modified by Sarsa evaluations.
*/

__device__
void evalHand(State *s) //, int *dcards, int *deck)
{
	int tid = threadIdx.x;
	HandLedger *h = &MasterLedger[tid];
	//dumpState(s,99);
//#ifdef NEVER
#if 1
	char dsum, issoft;
	//csum(s->ncards,s->cards,&s->issoft,&s->sum);
	if (s->sum>21) s->terminal = 1;
	if (s->terminal == 1) {
		if (s->sum < 22)
			if (verbose>0) {
			printf("iter %d: eH checking dealer. psum = %d\n", tid, s->sum);
			hipDeviceSynchronize();
			}
		dsumDeck(2,&issoft,&dsum);
		int handOver = false;
		if (DENOMVAL(dcards[tid][0])==1) {
			if (dsum==21) {
				if (verbose>1) printf("iter %d: Dealer has BJ!\n", tid);
				if (s->sum==21 && s->ncards==2) {
					if (verbose>1) printf("iter %d: Player has BJ and pushes!\n", tid);
					s->bank = 0;
				} else {
					if (verbose>1) printf("iter %d: Player has %d and loses!\n", tid, s->sum);
					s->bank = -(s->bet);
				}
				if (verbose>1) hipDeviceSynchronize();
				handOver = true;
			}
		}
		if (!handOver) {
			if (s->sum==21 && s->ncards==2) {
				if (verbose>1) {
					printf("iter %d: Player has BJ and wins!\n",tid);
					hipDeviceSynchronize();
				}
				s->bank = s->bet * 3 / 2;
				handOver = true;
			}
		}
		if (!handOver) {
			if (s->sum<22) {
				int ncards = 2;
				do {
					if (dsum < 17 || (dsum == 17 && issoft && HitSoft17)) {
						char nextC;
						dealDeck(&nextC);
						dcards[tid][ncards++] = nextC;
						dsumDeck(ncards,&issoft,&dsum);
						if (verbose>1) printf("iter %d: Dealer draws a %c.  Total is %d\n", tid, strcard[DENOM(nextC)],dsum);
					}
				}
				while (dsum < 17);
			}
			char *eval;
			if (s->sum > 21) {
				eval = (char*)"busts";
				s->bank = -(s->bet);
			} else if (s->sum < dsum && dsum < 22) {
				eval = (char*)"loses";
				s->bank = -(s->bet);
			} else if (s->sum == dsum) {
				eval = (char*)"pushes";
				s->bank = 0;
			} else {
				eval = (char*)"wins";
				s->bank = s->bet;
			}
			if (verbose>1) {
				char cardbuf[50];
				cardString(s->ncards,h->cards[s->handIndex],cardbuf);
				printf("iter %d: Hand %s = %d and %s $%d\n", tid, cardbuf, s->sum, eval, s->bank);
				hipDeviceSynchronize();
			}
		}
		/*
		if (s->parent != NULL) {
			printf("%lx (trm) passing %d up to %lx\n", s, s->bank, s->parent);
			hipDeviceSynchronize();
			s->parent->bank += s->bank;
		}
		*/
		//dumpState(s,0);
		return;
	}
#endif
	if (s->child1 != NULL) {
		State *ch = s->child1;
		//printf("follow %lx to ch1 %lx\n", s, ch);
		//dumpState(ch,1);
		evalHand(ch);//, dcards, deck);
		//printf("back from ch1 %lx to %lx\n", ch, s);
		//dumpState(ch,1);
		//dumpState(s,2);
		//if (s->child1->terminal)
		s->bank += ch->bank;
		//printf("%lx retrieves %d, now %d\n", s, ch->bank, s->bank);
	}
	/*
	if (s->parent != NULL) {
		//printf("%lx passing %d up to %lx\n", s, s->contrib, s->parent);
		s->parent->contrib += s->contrib;
	}
	hipDeviceSynchronize();
	if (s->child2 != NULL) { // hand was split into two
		//s->child2->dump(2, tid);
		evalHand(s->child2, dcards, deck);
		//if (s->child2->terminal) s->contrib += s->child2->contrib;
	}
	*/
}
