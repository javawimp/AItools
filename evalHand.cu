#include "includes.h"
#include "externs.h"
#include "bjState.h"

__device__ void evalHand(BJState *s, short *dcards, int *deck)
{
	//printf("adr %08lx\n",(unsigned long)s);
	//hipDeviceSynchronize();
	//printf("into eH nc %d term %d adr %08lx\n",s->ncards,s->terminal,(unsigned long)s);
	//hipDeviceSynchronize();
	if (s->child1 != NULL) {
        evalHand(s->child1, dcards, deck);
        if (s->child1->terminal) s->contrib = s->child1->contrib;
    }
    if (s->child2 != NULL) {
        evalHand(s->child2, dcards, deck);
        if (s->child1->terminal) s->contrib += s->child2->contrib;
    }
    if (s->terminal==0) {
    	//printf("eH bye\n");
    	//hipDeviceSynchronize();
    	return;
	}
	printf("eH checking dealer. psum = %d\n",s->sum);
	hipDeviceSynchronize();
    short dsum;
    bool issoft;
    if (s->sum<22) {
    	sum(2,dcards,&issoft,&dsum);
    	short ncards = 2;
        do {
            if (dsum < 17 || (dsum == 17 && issoft && HitSoft17)) {
            	short nextC;
            	nextCard(deck,&nextC);
				dcards[ncards++] = nextC;
		       	sum(ncards,dcards,&issoft,&dsum);
				if (verbose>1) printf("Dealer draws a %c.  Total is %d\n",strcard[DENOM(nextC)],dsum);
            }
        }
        while (dsum < 17);
    }
    char *eval;
    if (s->sum > 21) {
        eval = (char*)"busts";
        s->contrib = -s->bet;
    } else if (s->sum < dsum && dsum < 22) {
        eval = (char*)"loses";
        s->contrib = -s->bet;
    } else if (s->sum == dsum) {
        eval = (char*)"pushes";
        s->contrib = 0;
    } else {
        eval = (char*)"wins";
        if (s->sum == 21 && (s->ncards) == 2) {
            eval = (char*)"wins by BJ";
            s->contrib = s->bet * 3 / 2;
        } else s->contrib = s->bet;
    }
    if (verbose>1) {
    	char cardbuf[50];
    	cardString(s->ncards,s->cards,cardbuf);
        printf("Hand %s = %d and %s $%d\n", cardbuf, s->sum, eval, s->contrib);
    }
    //terminalState = s;
}
