#include "includes.h"
#include "externs.h"
#include "State.h"

// pick an action consistent with a state node's flags
__device__ void chooseAction(State *s,char *amax)
{
	int tid = threadIdx.x;
	*amax = AC_STAND;
	if (s->terminal == 1) return;
	for (;;) {
		*amax = (char) (hiprand_uniform(&randstate[tid]) * NB_ACTIONS);
		if (*amax == AC_DOUBLE && (s->flags & OK_DOUBLE) == 0) continue;
		if (*amax == AC_SPLIT && (s->flags & OK_SPLIT) == 0) continue;
		break;
	}
	//printf("choosing %d (%s)\n",* amax, choice[*amax]);
}

// pick an action consistent with a state node's flags using the table
__device__ void chooseSarsaAction(State *s, char *amax)
{
	*amax = AC_STAND;
	if (s->terminal == 1) return;
	chooseAction(s,amax);
        //if (Rand.randf() < epsilon) return amax;
        //*amax = 0;
        double qmax,q;
	Qget(s, *amax, &qmax);
        for (int i = 1; i < NB_ACTIONS; i++) {
            if (i == AC_DOUBLE && (s->flags & OK_DOUBLE) == 0) continue;
            if (i == AC_SPLIT && (s->flags & OK_SPLIT) == 0) continue;
	    Qget(s, i, &q);
            if (q > qmax) {
                Qget(s, i, &qmax);
                *amax = i;
            }
        }
	//printf("(Sarsa) choosing %d (%s)\n",* amax, choice[*amax]);
}

// recursively gen the game tree under this state node
__device__ void playHand(State *s)
{
	int tid = threadIdx.x;
	HandLedger *h = &MasterLedger[tid];
	if (s->terminal==1) {
		//dumpState(s,6);
		return;
	}
	char action;
	chooseSarsaAction(s,&action); // no sarsa learning
	s->action = action;
	//dumpState(s,5);
	hipDeviceSynchronize();
	if (action == AC_STAND) {
		s->terminal = 1;
		return;
	}

	char hix = s->handIndex;
	State *next;
	if (action == AC_HIT || action == AC_DOUBLE) {
		char nextC;
		dealDeck(&nextC);
		char cix = h->ncards[hix];
		h->cards[hix][cix] = nextC;
		h->ncards[hix] += 1;
		//if (verbose>0) printf("iter %d: for card [%d]%d Player draws a%s\n", tid, hix, h->ncards[hix], cardName[DENOM(nextC)]);
		allocateState(&next);
		s->child1 = next;
		//initFromParentState(s, next);
		next->parent = s;
		next->handIndex = hix;
		next->bet = s->bet;
		next->dealer = s->dealer;
		next->ncards = h->ncards[hix];
		psumDeck(hix,&next->issoft,&next->sum);
		if (next->sum > 21) next->terminal = 1;
	}
	if (verbose>0) {
		char cardbuf[50];
		cardString(next->ncards,h->cards[hix],cardbuf);
		printf("iter %d: [next card] nc=%d %s sum=%d\n",
			tid, next->ncards, cardbuf, next->sum);
	}

	if (s->child1 != NULL) playHand(s->child1);
}
