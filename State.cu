#include "includes.h"
#include "externs.h"

__device__ void clearState()
{
	int tid = threadIdx.x;
	union un u;
	u.hlp = &MasterLedger[tid];
	//printf("clear ledger [%d] at %lx\n",tid,u.hlp);
	for (int i=0;i<HL_ISIZE;++i) u.g[i] = 0;
}

__device__ void allocateState(State **p)
{
	int tid = threadIdx.x;
	HandLedger *h = &MasterLedger[tid];
	struct State *b = &h->bjstate[h->nextBlock++];
	//printf("[%d] allocated block at %lx\n",tid,b);
	*p = b;
}

__device__ void dumpState(State *p, int id)
{
	int tid = threadIdx.x;
	HandLedger *h = &MasterLedger[tid];
	printf("[%d][%lx] iter %d: act %d(%s) bet %d bank %d du %d nh %d nc %d sum %d fl %d t %d pr %lx ch1 %lx ch2 %lx\n",
		id,p,tid,
		p->action,choice[p->action],p->bet,p->bank,p->dealer,h->nhands,p->ncards,p->sum,p->flags,p->terminal,
		(long)p->parent,(long)p->child1,(long)p->child2);
	for (int nh=0; nh<h->nhands; ++nh) {
		for (int nc=0; nc<h->ncards[nh]; ++nc)
			printf("[%lx,%d,%d]",p,nc,DENOM(h->cards[nh][nc]));
		printf("\n");
	}
	hipDeviceSynchronize();
}
