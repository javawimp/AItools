#include "hip/hip_runtime.h"
#include "includes.h"
#include "externs.h"
#include "State.h"

__device__ void checkForBJs(char psum, char dsum, int *handOver)
{
	int tid = threadIdx.x;
	*handOver = 0;
	if ((DENOMVAL(dcards[tid][0])==1 || DENOMVAL(dcards[tid][1])==1) && dsum==21) *handOver = true;
	if (psum==21) *handOver = 1;
}

__global__ void doTrial()
{
	int tid = threadIdx.x;
	HandLedger *h = &MasterLedger[tid];

	//hipError_t st;
	//int *deck;
	char issoft, psum, dsum, dc;

	clearState();
	initializeDeck();
	shuffleDeck();
	//dealDeck(&h->cards[0][h->ncards[0]++]);
	//dealDeck(&h->cards[0][h->ncards[0]++]);
		char nextC;
		dealDeck(&nextC);
		h->cards[0][0] = nextC;
		dealDeck(&nextC);
		h->cards[0][1] = nextC;
		h->ncards[0] = 2;

	h->nhands = 1;
	dealDeck(&dcards[tid][0]);
	dealDeck(&dcards[tid][1]);
	dc=2;
	dsumDeck(dc,&issoft,&dsum);
	psumDeck(0,&issoft,&psum);
	if (verbose>9) {
		printf("iter %d: d %c %c   p %c %c = %d\n",tid,
			strcard[DENOM(dcards[tid][0])],strcard[DENOM(dcards[tid][1])],
			strcard[DENOM(h->cards[0][0])],strcard[DENOM(h->cards[0][1])],psum);
		hipDeviceSynchronize();
	}

	if (verbose>99) {
		printf("iter %d: Dealer has a%s showing\n", tid, cardName[DENOM(dcards[tid][0])]);
		hipDeviceSynchronize();
	}
	State *initialState;
	allocateState(&initialState);
	initialState->handIndex = 0;
	initialState->bet = 2;
	initialState->dealer = DENOMVAL(dcards[tid][0]);
	initialState->sum = psum;
	initialState->flags = OK_DOUBLE;
	initialState->issoft = issoft;
	initialState->ncards = 2;
	if (issoft) initialState->flags |= OK_SOFT; // make sure player is summed after dealer
	int handOver;
	checkForBJs(psum, dsum, &handOver);
	if (handOver==0) playHand(initialState);
	else initialState->terminal = 1;
	//for (State *p = initialState; p!=NULL; p = p->child1) printf("x%08lx . %d . %d . %d . %d . %d\n",
	//	(unsigned long)p,p->action,p->bet,p->bank,p->ncards,p->sum);
	//printf("------\n");
	evalHand(initialState);//, dcards, deck);
	if (initialState->child1) initialState->bank = initialState->child1->bank;
	//printf("%lx retrieves %d, now %d\n", initialState, initialState->child1->bank, initialState->bank);
	//dumpState(initialState, -1);
	atomicAdd(&bankroll,initialState->bank);
	if (verbose>-1) {
		printf("iter %d: Player hand $%d  total BR %d\n", tid, initialState->bank, bankroll);
	}
	updateSarsa(initialState);
}

int main(int argc, char *argv[]) {
	printf("welcome to cuda bj with sarsa!\n");
	int seed = 16;
	int iter = 1;
	int rounds = 1;
	for (int i=1;i<argc;++i) {
		if (!strcmp(argv[i],"-i")) iter = atoi(argv[++i]);
		if (!strcmp(argv[i],"-s")) seed = atoi(argv[++i]);
		if (!strcmp(argv[i],"-r")) rounds = atoi(argv[++i]);
	}
	#ifdef RESOURCE_CHECK
		size_t size;
		hipDeviceGetLimit(&size,hipLimitStackSize);
		printf("stack size %ld\n",size);
		hipError_t err = hipDeviceSetLimit(hipLimitStackSize, 3000);
		if (err) printf("ss set error %d\n",err);
		hipDeviceGetLimit(&size,hipLimitPrintfFifoSize);
		printf("print fifo size %ld\n",size);
		//err = hipDeviceSetLimit(hipLimitPrintfFifoSize, 2000000);
		//if (err) printf("fifo set error %d\n",err);
	#endif
	printf("sizeof(HandLedger) = %ld (%04lx)\n",sizeof(HandLedger),sizeof(HandLedger));
	struct HandLedger* d;
	hipMalloc((void**)&d, iter*sizeof(HandLedger));
	double* table;
	hipMalloc((void**)&table, (NB_SUMS * NB_DLR * NB_FLAGS*2 * NB_ACTIONS)*sizeof(double));

	initbj<<< 1, iter >>>(seed,d,table);
	hipDeviceSynchronize();
	printf("ready to play!\n");
	for (int round=0;round<rounds;++round) {
		doTrial<<< 1, iter >>>();
		hipDeviceSynchronize();
	}
	hipFree(d);
	hipFree(table);
	printf("wasn't that fun?!!\n");
} 
