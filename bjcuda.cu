#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>

#define DENOM(x) (((x-1)%13)+1)
#define DENOMV(x) (DENOM(x)>10?10:DENOM(x))
#define MAXCARDS 16 // if two decks, else 12

__shared__ int verbose;

__device__ const char *strcard = "?A23456789TJQK";
__shared__ int mutex;
__device__ void lock(int* mutex) {
  // capture lock when mutex = 0.
  // we will break out of the loop after mutex gets reset
  while (atomicCAS(mutex, 0, 1) != 0);
}
__device__ void unlock(int* mutex) {
  atomicExch(mutex, 0);
}

__device__ hiprandState state[256];

__global__ void initbj() {
	int tid = threadIdx.x;
	if (tid==0) {
		verbose = 1;
		mutex = 0;
	}
	hiprand_init(tid, 0, tid, &state[tid]);
}

__device__ void newDeck(int **deck,hipError_t *st) {
	int tid = threadIdx.x;
	if (verbose>2) printf("iter %d: shuffling\n",tid);
	unsigned int RAND;
	int temp[53];
	int *tdeck;
	*st = hipMalloc((void**)&tdeck, 53 * sizeof(int));
	if (*st != hipSuccess) {
		// compile with -rdc=true
		printf("cannot allocate deck, err = %d\n",*st);
		return;
	}
	int x = 0;
	for (int j=1; j<53; ++j) tdeck[x++] = j;
	tdeck[52] = 0;
	for (int newpos = 51; newpos >= 0; newpos--) {
	RAND = hiprand(&state[tid]);
	int oldpos = RAND % (newpos+1);
	temp[newpos] = tdeck[oldpos];
	for (int j = oldpos; j < newpos; j++) tdeck[j] = tdeck[j+1];
	}
	for (int i = 0; i < 52; ++i) tdeck[i] = temp[i];
/*
	// test the shuffle
	//lock(&mutex);
	printf("tid %d: ",threadIdx.x);
	for (int j = 0; j < 52; ++j) printf("%2d ",tdeck[j]);
	printf("\n");
	hipDeviceSynchronize();
	//unlock(&mutex);
*/
	*deck = tdeck;
}

__device__ void nextCard(int *deck, int *dealt) {
	int *card = deck;
	while (*card==-1) ++card;
	*dealt = *card;
	*card = -1;
	//printf("card %2d\n",*dealt);
}

__device__ void sum(int ncards, int *hand, int *result) {
	int sum = 0;
	int issoft = 0;
	for (int j=0; j<ncards; ++j) {
		int denom = DENOMV(hand[j]);
		if (denom == 1) {
			denom = 11;
			++issoft;
		}
		sum += denom;
	}
	while (sum > 21 && issoft > 0) {
		sum -= 10;
		--issoft;
	}
	*result = sum;
}

/*{
deal 2 cards to player
deal 2 cards to dealer
is dealer upcard ace?
	yes: dealer bj?
		yes: player bj?
			yes: push; return
			no: player loses; return
		no: playout()
	no: player bj?
		yes: player wins; return
		no: playout()

playout()
	action
	stand:
		dealer()
	if (psum<=21) hit:
		playout()
	if (ncards=2)
	double:
		draw()
		dealer()

dealer()
	if (psum>21) player loses; return
	if (player bj) player wins; return
	if (dsum>17) compare; return
	if (dsum>21 && soft) ddraw(); dealer()
	ddraw(); dealer()
}*/

__device__ void playout(int pc,int dc,int *pcards,int *dcards,int *deck,bool *done) {
	int tid = threadIdx.x;
	char buf[100];
	for (int i=0;i<pc;++i) { buf[i*2] = ' '; buf[i*2+1] = strcard[DENOMV(pcards[i])]; }
	buf[2*pc]=0;
	if (verbose>1) printf("tid %d: d shows %c p has%s\n",tid,strcard[DENOMV(dcards[0])],buf);
	int RAND = hiprand(&state[tid]);
	int psum;
	sum(pc,pcards,&psum);
	int action = RAND & 3;
	if (psum > 21) action = 0;
	if (pc > 2 && action == 2) action = 3;
	if (verbose>1) printf("iter %d: Player has %d action = %d\n",tid,psum,action);
	if (action == 0) {
		//printf("iter %d: done\n",tid);
		*done = true;
		return;
	}
	if (action == 2) {
		//printf("iter %d: doubling\n",tid);
		++pc;
		nextCard(deck,&pcards[pc-1]);
		if (verbose>1) printf("iter %d: Player doubles down and draws a %c\n",tid,strcard[DENOMV(pcards[pc-1])]);
		*done = true;
		return;
	}
	++pc;
	nextCard(deck,&pcards[pc-1]);
	playout(pc,dc,pcards,dcards,deck,done);
}

#define MAXITER 255

//__global__ void dealHand(int pc,int dc,int *pcards,int *dcards,int *deck) {
__global__ void dealHand() {
	int tid = threadIdx.x;
	if (tid>MAXITER) return;
	if (verbose>2) printf("iteration %d\n",tid);
	int *mydeck,psum,dsum,pc,dc,*pcards,*dcards;
	bool done = false;

	hipError_t st;
	newDeck(&mydeck,&st);
	if (st) return;
        hipMalloc((void**)&pcards, MAXCARDS * sizeof(int));
        hipMalloc((void**)&dcards, MAXCARDS * sizeof(int));
        nextCard(mydeck,&pcards[0]);
        nextCard(mydeck,&pcards[1]);
        pc=2;
        nextCard(mydeck,&dcards[0]);
        nextCard(mydeck,&dcards[1]);
        dc=2;
	if (verbose>1) printf("iter %d: p %c %c d %c %c\n",tid,
		strcard[DENOMV(pcards[0])],strcard[DENOMV(pcards[1])],
		strcard[DENOMV(dcards[0])],strcard[DENOMV(dcards[1])]);
	hipDeviceSynchronize();
	sum(pc,pcards,&psum);
	sum(dc,dcards,&dsum);
	if (DENOMV(dcards[0])==1) {
		if (dsum==21) {
			if (verbose>1)printf("iter %d: Dealer has BJ!\n",tid);
			if (psum==21) {
				if (verbose>1) printf("iter %d: Player has BJ!\n",tid);
				if (verbose>1) printf("iter %d: Player pushes!\n",tid);
			} else {
				if (verbose>1) printf("iter %d: Player has %d\n",tid,psum);
				if (verbose>1) printf("iter %d: Player loses!\n",tid);
			}
			done = true;
		}
	}
	if (!done) {
		if (psum==21) {
			if (verbose>1) printf("iter %d: Player has BJ!\n",tid);
			if (verbose>1) printf("iter %d: Player wins!\n",tid);
			done = true;
		}
	}

	if (!done) {
		if (verbose>2) printf("tid %d: more to come. p = %d d = %d\n",tid,psum,dsum);
		playout(pc,dc,pcards,dcards,mydeck,&done);
	}
	hipFree(pcards);
	hipFree(dcards);
	hipFree(mydeck);
}

int main(int argc, char *argv[]) {
	//hipDeviceReset();
	printf("welcome to cuda bj!\n");
	initbj<<<1,MAXITER+1>>>();
	hipDeviceSynchronize();
	printf("ready to play!\n");
	for (int round=0;round<20;++round) {
		dealHand<<<1,MAXITER+1>>>();
		hipDeviceSynchronize();
	}
	//dealHand<<<1,MAXITER+1>>>();
	//hipDeviceSynchronize();
	printf("was that fun?\n");
} 
