#include "hip/hip_runtime.h"
#include "includes.h"
#include "externs.h"

__device__ const char *strcard = "?A23456789TJQK";

__device__ void playout(int pc,int dc,int *pcards,int *dcards,int *deck,bool *done) {
	int tid = threadIdx.x;
	char buf[100];
	for (int i=0;i<pc;++i) { buf[i*2] = ' '; buf[i*2+1] = strcard[DENOMV(pcards[i])]; }
	buf[2*pc]=0;
	if (verbose>1) printf("tid %d: d shows %c p has%s\n",tid,strcard[DENOMV(dcards[0])],buf);
	int RAND = hiprand(&state[tid]);
	int psum;
	sum(pc,pcards,&psum);
	int action = RAND & 3;
	if (psum > 21) action = 0;
	if (pc > 2 && action == 2) action = 3;
	if (verbose>1) printf("iter %d: Player has %d action = %d\n",tid,psum,action);
	if (action == 0) {
		//printf("iter %d: done\n",tid);
		*done = true;
		return;
	}
	if (action == 2) {
		//printf("iter %d: doubling\n",tid);
		++pc;
		nextCard(deck,&pcards[pc-1]);
		if (verbose>1) printf("iter %d: Player doubles down and draws a %c\n",tid,strcard[DENOMV(pcards[pc-1])]);
		*done = true;
		return;
	}
	++pc;
	nextCard(deck,&pcards[pc-1]);
	playout(pc,dc,pcards,dcards,deck,done);
}

__global__ void dealHand() {
	int tid = threadIdx.x;
	if (tid>MAXITER) return;
	if (verbose>2) printf("iteration %d\n",tid);
	int *mydeck,psum,dsum,pc,dc,*pcards,*dcards;
	bool done = false;

	hipError_t st;
	newDeck(&mydeck,&st);
	if (st) return;
        hipMalloc((void**)&pcards, MAXCARDS * sizeof(int));
        hipMalloc((void**)&dcards, MAXCARDS * sizeof(int));
        nextCard(mydeck,&pcards[0]);
        nextCard(mydeck,&pcards[1]);
        pc=2;
        nextCard(mydeck,&dcards[0]);
        nextCard(mydeck,&dcards[1]);
        dc=2;
	if (verbose>1) printf("iter %d: p %c %c d %c %c\n",tid,
		strcard[DENOMV(pcards[0])],strcard[DENOMV(pcards[1])],
		strcard[DENOMV(dcards[0])],strcard[DENOMV(dcards[1])]);
	hipDeviceSynchronize();
	sum(pc,pcards,&psum);
	sum(dc,dcards,&dsum);
	if (DENOMV(dcards[0])==1) {
		if (dsum==21) {
			if (verbose>1)printf("iter %d: Dealer has BJ!\n",tid);
			if (psum==21) {
				if (verbose>1) printf("iter %d: Player has BJ!\n",tid);
				if (verbose>1) printf("iter %d: Player pushes!\n",tid);
			} else {
				if (verbose>1) printf("iter %d: Player has %d\n",tid,psum);
				if (verbose>1) printf("iter %d: Player loses!\n",tid);
			}
			done = true;
		}
	}
	if (!done) {
		if (psum==21) {
			if (verbose>1) printf("iter %d: Player has BJ!\n",tid);
			if (verbose>1) printf("iter %d: Player wins!\n",tid);
			done = true;
		}
	}

	if (!done) {
		if (verbose>2) printf("tid %d: more to come. p = %d d = %d\n",tid,psum,dsum);
		playout(pc,dc,pcards,dcards,mydeck,&done);
	}
	hipFree(pcards);
	hipFree(dcards);
	hipFree(mydeck);
}

int main(int argc, char *argv[]) {
	printf("welcome to cuda bj!\n");
	initbj<<<1,MAXITER+1>>>();
	hipDeviceSynchronize();
	printf("ready to play!\n");
	for (int round=0;round<1;++round) {
		dealHand<<<1,MAXITER+1>>>();
		hipDeviceSynchronize();
	}
	printf("was that fun?\n");
} 
