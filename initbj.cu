#include "hip/hip_runtime.h"
#include "includes.h"
#include "State.h"

__shared__ int verbose;
__shared__ const char *strcard;
__shared__ int bankroll;
__shared__ const char *choice[4];
__shared__ const char *cardName[14];
__device__ hiprandState randstate[1024];
__device__ char dcards[1024][MAXCARDS];
__shared__ HandLedger *MasterLedger;
__shared__ double Sarsa_alpha;
__shared__ double Sarsa_gamma;
//__shared__ double table[NB_SUMS][NB_DLR][NB_FLAGS * 2][NB_ACTIONS];
__shared__ double *table;

__global__ void initbj(int seed, HandLedger *hh, double *t) {
	int tid = threadIdx.x;
	if (tid==0) {
		MasterLedger = hh;
		table = t;
		verbose = 0;
		strcard = "?A23456789TJQK";
		const char *ch[4] = {"STAND","HIT","DOUBLE","SPLIT"};
		for (int i=0;i<4;++i) choice[i] = ch[i];
		const char *cn[14] = { "","n A"," 2"," 3"," 4"," 5"," 6"," 7","n 8"," 9"," 10"," J"," Q"," K" };
		for (int i=0;i<14;++i) cardName[i] = cn[i];
		bankroll = 0;
		Sarsa_alpha = 0.1;
		Sarsa_gamma = 0.9;
	}
	hiprand_init(tid+seed, 0, tid, &randstate[tid]);
}
