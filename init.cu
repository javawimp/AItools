#include "hip/hip_runtime.h"
#include "includes.h"

__shared__ int verbose;
__shared__ const char *strcard;

__shared__ int mutex;
__device__ void lock(int* mutex) {
	// capture lock when mutex = 0.
	// we will break out of the loop after mutex gets reset
	while (atomicCAS(mutex, 0, 1) != 0);
}
__device__ void unlock(int* mutex) {
	atomicExch(mutex, 0);
}

__device__ hiprandState state[MAXITER];

__global__ void initbj(int seed) {
	int tid = threadIdx.x;
	if (tid==0) {
		verbose = 2;
		mutex = 0;
		strcard = "?A23456789TJQK";
	}
	hiprand_init(tid+seed, 0, tid, &state[tid]);
}
