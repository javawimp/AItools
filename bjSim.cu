#include "hip/hip_runtime.h"
#include "includes.h"
#include "externs.h"
#include "bjState.h"

__global__ void doTrial()
{
	int tid = threadIdx.x;
	const char *dlrcard[14] = { "","n A"," 2"," 3"," 4"," 5"," 6"," 7","n 8"," 9"," 10"," J"," Q"," K" };

	hipError_t st;
	int *deck;
	bool issoft;
	short psum,dsum,pc,dc,*pcards,*dcards;
	BJState *terminalState = NULL;

	newDeck(&deck,&st);
    hipMalloc((void**)&pcards, MAXCARDS * sizeof(int));
    hipMalloc((void**)&dcards, MAXCARDS * sizeof(int));
    nextCard(deck,&pcards[0]);
    nextCard(deck,&pcards[1]);
    pc=2;
    nextCard(deck,&dcards[0]);
    nextCard(deck,&dcards[1]);
    dc=2;
	sum(pc,pcards,&issoft,&psum);
	sum(dc,dcards,&issoft,&dsum);
	if (verbose) {
		printf("iter %d: d %c %c p %c %c = %d\n",tid,
			strcard[DENOMV(dcards[0])],strcard[DENOMV(dcards[1])],
			strcard[DENOMV(pcards[0])],strcard[DENOMV(pcards[1])],psum);
		hipDeviceSynchronize();
	}

	if (verbose) {
		printf("Dealer has a%s showing\n", dlrcard[DENOM(dcards[0])]);
		hipDeviceSynchronize();
	}
	BJState initialState = BJState();
	initialState.bet = 2;
	initialState.dealerUp = DENOMV(dcards[0]);
	initialState.ncards = 2;
	initialState.cards[0] = pcards[0];
	initialState.cards[1] = pcards[1];
	initialState.sum = psum;
	initialState.flags = OK_DOUBLE | OK_SOFT;
	
	playHand(&initialState,deck);
/*
	master = learner.nextState(initialState, BJSarsa.HIT, d.hand[1][1]);
	initialState.child1 = master;
	learner.playHand(master);
	boolean noDlrBJ = game.finish(iteration);
	if (noDlrBJ) {
		learner.evalHands(master, game.dsum);
		game.bankroll += terminalState.contrib;
		if (verbose) System.out.printf("Player $%d  BR %d $/hand = %.03f\n", terminalState.contrib,game.bankroll,(float)game.bankroll/(float)iteration);
		learner.updateState(master);
	}
*/
	delete &initialState;
	hipFree(pcards);
	hipFree(dcards);
	hipFree(deck);
}

int main(int argc, char *argv[]) {
	printf("welcome to cuda bj with sarsa!\n");
	int seed = 16;
	if (argc>1) seed = atoi(argv[1]);
	initbj<<<1,1>>>(seed);
	hipDeviceSynchronize();
	printf("ready to play!\n");
	for (int round=0;round<1;++round) {
		doTrial<<<1,1>>>();
		hipDeviceSynchronize();
	}
	printf("wasn't that fun?!!\n");
} 
