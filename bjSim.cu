#include "hip/hip_runtime.h"
#include "includes.h"
#include "externs.h"
#include "bjState.h"

__device__ void checkForBJs(short psum,short dsum,short *pcards,short *dcards,bool *handOver)
{
	int tid = threadIdx.x;
	*handOver = false;
	if (DENOMV(dcards[0])==1) {
		if (dsum==21) {
			if (verbose>1)printf("iter %d: Dealer has BJ!\n",tid);
			if (psum==21) {
				if (verbose>1) printf("iter %d: Player has BJ!\n",tid);
				if (verbose>1) printf("iter %d: Player pushes!\n",tid);
			} else {
				if (verbose>1) printf("iter %d: Player has %d\n",tid,psum);
				if (verbose>1) printf("iter %d: Player loses!\n",tid);
			}
			*handOver = true;
		}
	}
	if (!*handOver) {
		if (psum==21) {
			if (verbose>1) printf("iter %d: Player has BJ!\n",tid);
			if (verbose>1) printf("iter %d: Player wins!\n",tid);
			*handOver = true;
		}
	}
	hipDeviceSynchronize();
}

__global__ void doTrial()
{
	int tid = threadIdx.x;
	const char *dlrcard[14] = { "","n A"," 2"," 3"," 4"," 5"," 6"," 7","n 8"," 9"," 10"," J"," Q"," K" };

	hipError_t st;
	int *deck;
	bool issoft;
	short psum,dsum,pc,dc;
	short pcards[MAXCARDS],dcards[MAXCARDS];
	BJState *terminalState = NULL;

	newDeck(&deck,&st);
    nextCard(deck,&pcards[0]);
    nextCard(deck,&pcards[1]);
    pc=2;
    nextCard(deck,&dcards[0]);
    nextCard(deck,&dcards[1]);
    dc=2;
	sum(pc,pcards,&issoft,&psum);
	sum(dc,dcards,&issoft,&dsum);
	if (verbose>1) {
		printf("iter %d: d %c %c   p %c %c = %d\n",tid,
			strcard[DENOM(dcards[0])],strcard[DENOM(dcards[1])],
			strcard[DENOM(pcards[0])],strcard[DENOM(pcards[1])],psum);
		hipDeviceSynchronize();
	}

	if (verbose>1) {
		printf("Dealer has a%s showing\n", dlrcard[DENOM(dcards[0])]);
		hipDeviceSynchronize();
	}
	BJState initialState = BJState();
	initialState.bet = 2;
	initialState.dealerUp = DENOMV(dcards[0]);
	initialState.ncards = 2;
	initialState.cards[0] = pcards[0];
	initialState.cards[1] = pcards[1];
	initialState.sum = psum;
	initialState.flags = OK_DOUBLE | OK_SOFT;
	bool handOver;
	checkForBJs(psum,dsum,pcards,dcards,&handOver);
	if (!handOver) {
		playHand(&initialState,deck);
		//printf("ok, now eval...\n");
		evalHand(&initialState, dcards, deck);
/*
		game.bankroll += terminalState.contrib;
		if (verbose) System.out.printf("Player $%d  BR %d $/hand = %.03f\n", terminalState.contrib,game.bankroll,(float)game.bankroll/(float)iteration);
		learner.updateState(master);
*/
	}
	delete &initialState;
	hipFree(deck);
}

int main(int argc, char *argv[]) {
	printf("welcome to cuda bj with sarsa!\n");
	int seed = 16;
	int iter = 1;
	for (int i=1;i<argc;++i) {
		if (!strcmp(argv[i],"-i")) iter = atoi(argv[++i]);
		if (!strcmp(argv[i],"-s")) seed = atoi(argv[++i]);
	}
	initbj<<<1,iter>>>(seed);
	hipDeviceSynchronize();
	printf("ready to play!\n");
	for (int round=0;round<1;++round) {
		doTrial<<<1,iter>>>();
		hipDeviceSynchronize();
	}
	printf("wasn't that fun?!!\n");
} 
